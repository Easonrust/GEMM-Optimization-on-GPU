#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code
#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"

using namespace std;

#include <stdio.h>

__global__ void matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B)
{
    __shared__ _DOUBLE_ As[MC][KC];
    __shared__ _DOUBLE_ Bs[KC][NC];
    // Load fraction of shared memory into register
    _DOUBLE_ sAs[TM];
    _DOUBLE_ sBs[TN];
    _DOUBLE_ Cs[TM][TN] = {0};

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int by = blockIdx.y;
    int bx = blockIdx.x;
    int I = by*MC + ty; 
    int J = bx*NC + tx;

    int end = N/KC + (N%KC!=0);

#pragma unroll 
    for(int kk=0; kk<end; kk++){
#pragma unroll
        for(int i=0; i<TM; i++){
#pragma unroll
            for(int j=0; j<KC/BLOCKDIM_X; j++){
                int ii = I + i*BLOCKDIM_Y;
                int jj = kk*KC+tx + j*BLOCKDIM_X;
                As[ty + i*BLOCKDIM_Y][tx + j*BLOCKDIM_X] = ii<N && jj<N ? A[ii*N + jj] : 0;
            }
        }
#pragma unroll
        for(int i=0; i<KC/BLOCKDIM_Y; i++){
#pragma unroll
            for(int j=0; j<TN; j++){
                int ii = kk*KC+ty + i*BLOCKDIM_Y;
                int jj = J + j*BLOCKDIM_X;
                Bs[ty + i*BLOCKDIM_Y][tx + j*BLOCKDIM_X] = ii<N && jj<N ? B[ii*N + jj] : 0;
            }
        }
        __syncthreads();
#pragma unroll
        for (int k=0; k<KC; k++){
#pragma unroll
            for(int i=0; i<TM; i++){
                sAs[i] = As[ty + BLOCKDIM_Y*i][k];
            }
#pragma unroll
            for(int i=0; i<TN; i++){
                sBs[i] = Bs[k][tx + BLOCKDIM_X*i];
            }
#pragma unroll
            for(int i=0; i<TM; i++){
#pragma unroll
                for(int j=0; j<TN; j++){
                    Cs[i][j] += sAs[i]*sBs[j];
                }
            }
        }
        __syncthreads();
    }
#pragma unroll
    for(int i=0; i<TM; i++){
#pragma unroll
        for(int j=0; j<TN; j++){
            int ii = I + i*BLOCKDIM_Y;
            int jj = J + j*BLOCKDIM_X;
            if(ii<N && jj<N){
                C[ii*N + jj] = Cs[i][j];
            }
        }
    }
}

__global__ void shared_matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B)
{
    __shared__ _DOUBLE_ As[TW][TW];
    __shared__ _DOUBLE_ Bs[TW][TW];

    int ty = threadIdx.y;
    int tx = threadIdx.x;
    int by = blockIdx.y; 
    int bx = blockIdx.x;
    int I = by*TW + ty; 
    int J = bx*TW + tx;
    double Cij = 0;
    
    int A_idx;
    int B_idx;
    for (int kk=0; kk<N/TW; kk++){
        A_idx = I*N+kk*TW+tx;
        B_idx = (kk*TW+ty)*N + J;

        // padding
        if(A_idx<N*N){
            As[ty][tx] = A[A_idx];
        }else{
            As[ty][tx] = 0;
        }
        if(B_idx<N*N){
            Bs[ty][tx] = B[B_idx];
        }else{
            Bs[ty][tx] = 0;
        }

        __syncthreads();
        for (int k=0; k<TW; k++)
            Cij+= As[ty][k] * Bs[k][tx];
        __syncthreads();
    }
    if(I<N&&J<N){
        C[I*N + J] = Cij;
    }
}

__global__ void original_matMul(int N, _DOUBLE_ *C, _DOUBLE_ *A, _DOUBLE_ *B)
{

    int I = blockIdx.y * blockDim.y + threadIdx.y;
    int J = blockIdx.x * blockDim.x + threadIdx.x;

    if ((I < N) && (J < N))
    {
        _DOUBLE_ _c = 0;
        for (unsigned int k = 0; k < N; k++)
        {
            _DOUBLE_ a = A[I * N + k];
            _DOUBLE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}
